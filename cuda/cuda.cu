#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "numgen.c"
#include <stdbool.h>

__device__ bool is_prime_device(unsigned long int num) {
    if (num <= 1) return false;
    for (unsigned long int i = 2; i * i <= num; i++) {
        if (num % i == 0) return false;
    }
    return true;
}

__global__ void count_primes_parallel(unsigned long int *numbers, int *prime_count, int inputArgument) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < inputArgument) {
        if (is_prime_device(numbers[idx])) {
            atomicAdd(prime_count, 1); 
        }
    }
}

int main(int argc,char **argv) {
    Args ins__args;
    parseArgs(&ins__args, &argc, argv);
    long inputArgument = ins__args.arg; 

    unsigned long int *numbers = (unsigned long int*)malloc(inputArgument * sizeof(unsigned long int));
    numgen(inputArgument, numbers);

    struct timeval ins__tstart, ins__tstop;
    gettimeofday(&ins__tstart, NULL);  

    unsigned long int *dev_numbers;
    int *dev_prime_count;
    hipMalloc(&dev_numbers, inputArgument * sizeof(unsigned long int));
    hipMalloc(&dev_prime_count, sizeof(int));
    hipMemcpy(dev_numbers, numbers, inputArgument * sizeof(unsigned long int), hipMemcpyHostToDevice);
    hipMemset(dev_prime_count, 0, sizeof(int));

    int threadsPerBlock = 256;
    int blocks = (inputArgument + threadsPerBlock - 1) / threadsPerBlock; 
    count_primes_parallel<<<blocks, threadsPerBlock>>> (dev_numbers, dev_prime_count, inputArgument);

    int prime_count_host;
    hipMemcpy(&prime_count_host, dev_prime_count, sizeof(int), hipMemcpyDeviceToHost);

    gettimeofday(&ins__tstop, NULL);
    ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);

    printf("Number of prime numbers: %d\n", prime_count_host);

    hipFree(dev_numbers);
    hipFree(dev_prime_count); 
    free(numbers);

    return 0;
}
